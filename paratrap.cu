﻿#include<iostream>
#include<iomanip>
#include<math.h>
#include<chrono>
#include<fstream>
// CUDA includes
#include <hip/hip_runtime.h>


#include <hip/hip_runtime_api.h>

// Number of threads per block
int BLOCK_SIZE = 256;

// GPU code for computing the function values on the grid points
__global__ void f_eval(double* f, double h, int n)
{
	// thread number
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i <= n) {
		// set x value for i
		double x = i * h;

		// compute the function value at x
		f[i] = sqrt((exp(cos(pow(pow(pow(x, x), x), x)))));
	}
}


// Trapezoidal Rule in parallel
double parallelQuad(const double alpha, const double beta, const int N)
{
	std::cout << "Computing integral in parallel\n";
	std::ofstream myfile;
	myfile.open("results_time_f2.txt", std::ios_base::app);

	// Set timer start
	auto start = std::chrono::high_resolution_clock::now();

	// Compute interval size
	const double h = (beta - alpha) / N;

	// allocate host memory for function evaluations on grid
	const int ARRAY_BYTES = sizeof(double) * (N + 1);
	double * h_farray = new double [N + 1]; // limited to size < 1 million bc of stack size

	// allocate GPU memory for function evaluations on grid
	double* d_farray;
	hipMalloc(&d_farray, ARRAY_BYTES);
	hipMemset(d_farray, 0, ARRAY_BYTES);

	// compute number of blocks required and launch kernal for each point in the grid
	int num_blocks = (N / BLOCK_SIZE) + 1;
	f_eval <<<num_blocks, BLOCK_SIZE>>> (d_farray, h, N);

	// copy back the array from GPU memory, then free GPU memory
	hipMemcpy(h_farray, d_farray, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipFree(d_farray);

	// Compute approximation and remove memory allocation
	double intgrl = 0.5 * (h_farray[0] + h_farray[N]);
	for (int k = 1; k < N; k++) {
		intgrl += h_farray[k];
	}
	intgrl *= h;
	delete[] h_farray;

	// Set timer end, compute duration, write to file
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds> (end - start);
	myfile << duration.count() << ",";
	myfile.close();

	// Print results to console
	std::cout << "Integral is approximately: " << std::fixed << std::setprecision(16) << intgrl << "\n";
	std::cout << "Computed in: " << duration.count() << " (microseconds)\n\n";

	// Return intgrl value
	return intgrl;
}

// Richardson Extrapolation in parallel
double richardsonQuad(const double alpha, const double beta, const int N)
{
	std::cout << "Computing in parallel with Richardson Extrapolation\n";
	std::ofstream myfile;
	myfile.open("results_time_f2.txt", std::ios_base::app);

	// Set timer start
	auto start = std::chrono::high_resolution_clock::now();

	// Compute interval size and half interval size
	const double h = (beta - alpha) / N;
	const double h_upon2 = 0.5 * h;

	// allocate host memory for function evaluations on h_upon2 sized grid
	const int ARRAY_BYTES = sizeof(double) * (2 * N + 1);
	double* h_farray = new double[2 * N + 1];

	// allocate gpu memory for function evaluations on grid
	double* d_farray;
	hipMalloc(&d_farray, ARRAY_BYTES);
	hipMemset(d_farray, 0, ARRAY_BYTES);

	// compute number of blocks required and launch kernal for each point in the grid
	int num_blocks = (2 * N / BLOCK_SIZE) + 1;
	f_eval<<<num_blocks, BLOCK_SIZE >>>(d_farray, h_upon2, 2 * N);

	// copy back the array from GPU memory, then free GPU memory
	hipMemcpy(h_farray, d_farray, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipFree(d_farray);

	// Compute approximations on both grid levels and remove memory allocation
	double trap_h_upon2 = 0.5 * (h_farray[0] + h_farray[2 * N]);
	double trap_h = trap_h_upon2;
	for (int k = 1; k < 2 * N; k++) {
		trap_h_upon2 += h_farray[k];
		if (k % 2 == 0) {
			trap_h += h_farray[k];
		}
	}
	double intgrl = (4 / 3) * (h_upon2 * trap_h_upon2) - (1 / 3) * (h * trap_h);
	delete[] h_farray;

	// Set timer end, compute duration, write to file
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds> (end - start);
	myfile << duration.count() << ",";
	myfile.close();

	// Print results to console
	std::cout << "Integral is approximately: " << std::fixed << std::setprecision(16) << intgrl << "\n";
	std::cout << "Computed in: " << duration.count() << " (microseconds)\n\n";

	return intgrl;
}