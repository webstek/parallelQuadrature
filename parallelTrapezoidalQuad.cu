﻿#include<iostream>
#include<iomanip>
#include <math.h>
#include<chrono>

#include <hip/hip_runtime.h>


// Computes the integral: int_0^1{ exp(cos(x)) dx} using a uniform grid compound Trapezoidal parallel method
#define BLOCK_SIZE 256

// Endpoints and number of grid intervals
const double ALPHA = 0;
const double BETA = 1;
const int N = 1000;
const double h = (BETA - ALPHA) / N;

void print_array(double* array, int size);

// GPU code for computing the function values on the grid points
__global__ void double_f_eval(double* f, double h, int n)
{
	// thread number
	// TODO may need to change below formula to incorporate more threads than in 1 thread block
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i <= n) {
		// set x value for i
		double x = i * h;

		// compute the function value at x (for compount Trapezoidal rule) --> the function is f(x) = exp(cos(x)) in this case.
		f[i] = exp(cos(x));
	}
}


// CPU Code to compute approximation of the integral
int main(int argc, char** argv)
{
	std::cout << "Computing int_0^1 exp(cos(x)) in parallel on " << N+1 << " grid points...\n";

	// Set timer start
	auto start = std::chrono::high_resolution_clock::now();

	// allocate host memory for function evaluations on grid
	const int ARRAY_BYTES = sizeof(double) * (N + 1);
	double h_farray[N + 1]; // limited to size < 1 million bc of stack size
	// double *h_farray = (double*)malloc(ARRAY_BYTES);

	// allocate GPU memory for function evaluations on grid
	double* d_farray;
	hipMalloc(&d_farray, ARRAY_BYTES);
	hipMemset(d_farray, 0, ARRAY_BYTES);

	// compute number of blocks required and launch kernal for each point in the grid
	int num_blocks = (N / BLOCK_SIZE) + 1;
	double_f_eval <<<num_blocks, BLOCK_SIZE>>> (d_farray, h, N);

	// copy back the array from GPU memory, then free GPU memory
	hipMemcpy(h_farray, d_farray, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipFree(d_farray);

	// Compute approximation
	double intgrl = 0.5 * (h_farray[0] + h_farray[N]);
	for (int k = 1; k < N; k++) {
		intgrl += h_farray[k];
	}
	intgrl *= h;


	// Set timer end, computer duration
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds> (end - start);


	// Print results to console
	std::cout << "Integral is approximately: " << std::fixed << std::setprecision(20) << intgrl << "\n";
	std::cout << "Computed in: " << duration.count() << " (milliseconds)\n";

	return 0;
}

// Display's an Array
void print_array(double* array, int size)
{
	std::cout << "[";
	for (int i = 0; i < size-1; i++) { std::cout << array[i] <<", "; }
	std::cout << array[size-1] << "]\n";
}